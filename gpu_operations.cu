#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <iostream>
#include "gpu_operations.h"

using namespace std;

/*
	This file contains the following functions:

		MatrixMatrixMultGPU => C = A * B
		MatrixVectorMultGPU => y = A * x
		VectorAddGPU	    => w = u + v
		VectorDotGPU	    => c = u * v
*/

void MatrixMatrixMultGPU(float *A, int A_m, int A_n, float *B, int B_m, int B_n, float *C) {
/*
	This function computes:

		C = A * B		

	MatrixMatrixMultGPU takes in 7 parameters:
		A   - matrix A
		A_m - # of rows in A
		A_n - # of columns in A
		B   - matrix B
		B_m - # of rows in B
		B_n - # of columns in B
		C   - matrix C 	
*/	

	if (A_n != B_m) {
		cout << "Matrix/Matrix sizing error" << endl;
		C = NULL;
		return;
	}
	
	int A_size = A_m * A_n;
	int B_size = B_m * B_n;
	int C_size = A_m * B_n;
	float *d_A, *d_B, *d_C;

	hipMalloc(&d_A, A_size * sizeof(float));
	hipMalloc(&d_B, B_size * sizeof(float));
	hipMalloc(&d_C, C_size * sizeof(float));

	hipMemcpy(d_A, A, A_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, B_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, C_size * sizeof(float), hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	const float alpha = 1.0f;
	const float beta = 0.0f;
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
			A_m, B_n, A_n, &alpha, d_A, 
			A_m, d_B, A_n, &beta , d_C, A_n);

	hipblasDestroy(handle);
	
	hipMemcpy(C, d_C, C_size * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	
}

void MatrixVectorMultGPU(float *A, int A_m, int A_n, float *x, int x_m, float *y) {
/* 
	This function computes:
		
		y = Ax

	MatrixVectorMultGPU takes in 6 parameters:
		A   - matrix A
		A_m - # of rows in A
		A_n - # of columns in A
		x   - vector x
		x_n - # of elements in x
		y   - vector y			
*/

	if (A_n != x_m) {
		cout << "Matrix/Vector sizing error" << endl;
		y = NULL;
		return;
	}

	int A_size = A_m * A_n;
	float *d_A, *d_x, *d_y;
	
	hipMalloc(&d_A, A_size * sizeof(float));
	hipMalloc(&d_x, x_m 	* sizeof(float));
	hipMalloc(&d_y, A_m	* sizeof(float));	

	hipMemcpy(d_A, A, A_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, x_m    * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, x_m	  * sizeof(float), hipMemcpyHostToDevice);
	
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	const float alpha = 1.0f;
	const float beta = 0.0f;	
	hipblasSgemv(handle, HIPBLAS_OP_T, A_m, A_n, &alpha, d_A, A_m, d_x, 1, &beta, d_y, 1);
	
	hipMemcpy(y, d_y, A_m * sizeof(float), hipMemcpyDeviceToHost);
	
	hipblasDestroy(handle);

	hipFree(d_A);
	hipFree(d_x);
	hipFree(d_y);
}

__global__ void VectAdd(float *u, float *v, float *w, int n) {
	int i = threadIdx.x;
	if (i < n) {
		w[i] = u[i] + v[i];
	}
}

void VectorAddGPU(float *u, float *v, float *w, int n) {
/*
	This function computes:

		w = u + v

	VectorAddGPU takes in 4 parameters:
		u - vector u
		v - vector v
		w - vector w
		n - # of elements in u, v, w
*/	
	float memsize = n * sizeof(float);
	float *d_u, *d_v, *d_w;
	hipMalloc(&d_u, memsize);
	hipMalloc(&d_v, memsize);
	hipMalloc(&d_w, memsize);

	hipMemcpy(d_u, u, memsize, hipMemcpyHostToDevice);	
	hipMemcpy(d_v, v, memsize, hipMemcpyHostToDevice);	
	hipMemcpy(d_w, w, memsize, hipMemcpyHostToDevice);	

	VectAdd<<<1, n>>>(d_u, d_v, d_w, n);

	hipMemcpy(w, d_w, memsize, hipMemcpyDeviceToHost);

	hipFree(d_u);
	hipFree(d_v);
	hipFree(d_w);
}

void VectorDotGPU(float *u, float *v, float *c, int n) {
/*
	This function computes:
		
		c = u * v

	VectorDotGPU takes 4 parameters:
		u - vector u
		v - vector v
		c - scalar output
		n - # of elements in u, v

	Usage:
		VectorDotGPU(h_u, h_v, &h_c, n);
*/
	float memsize = n * sizeof(float);
	float *d_u, *d_v, *d_c;

	hipMalloc(&d_u, memsize);
	hipMalloc(&d_v, memsize);
	hipMalloc(&d_c, sizeof(float));

	hipMemcpy(d_u, u, memsize, hipMemcpyHostToDevice);
	hipMemcpy(d_v, v, memsize, hipMemcpyHostToDevice);	

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

	hipblasSdot(handle, n, 
			d_u, 1, 
			d_v, 1, 
			d_c);

	hipblasDestroy(handle);

	hipMemcpy(c, d_c, sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_u);
	hipFree(d_v);
	hipFree(d_c);
}

