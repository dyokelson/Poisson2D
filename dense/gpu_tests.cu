#include "gpu_tests.h"
#include <iostream>
#include "gpu_operations.h"

#define M 4
#define N 4

using namespace std;

void test_gpu_operations() {
    double c;
    double *A = (double *)malloc(N * M * sizeof(double)); double *d_A; hipMalloc(&d_A, N * M * sizeof(double));
    double *C = (double *)malloc(N * M * sizeof(double)); double *d_C; hipMalloc(&d_C, N * M * sizeof(double));
    double *u = (double *)malloc(N * sizeof(double));     double *d_u; hipMalloc(&d_u, N * sizeof(double));
    double *v = (double *)malloc(N * sizeof(double));     double *d_v; hipMalloc(&d_v, N * sizeof(double));
    double *w = (double *)malloc(N * sizeof(double));     double *d_w; hipMalloc(&d_w, N * sizeof(double));
    double *x = (double *)malloc(N * sizeof(double));     double *d_x; hipMalloc(&d_x, N * sizeof(double));
    double *y = (double *)malloc(N * sizeof(double));     double *d_y; hipMalloc(&d_y, N * sizeof(double));
    double *b = (double *)malloc(N * sizeof(double)); 


    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[(i * N) + j] = i;
            C[(i * N) + j] = 0;
        }
        u[i] = i;
        v[i] = i;
        w[i] = 0;
        x[i] = 0;
    }
    x[3] = 1;

    hipMemcpy(d_A, A, N * M * sizeof(double), hipMemcpyHostToDevice);    
    hipMemcpy(d_u, u, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);

    cout << "Vector Add:" << endl;
    VectorAddGPU(d_u, d_v, 1.0, d_w, N);
    hipMemcpy(w, d_w, N * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        cout << w[i] << endl;
    }

    cout << endl << "Vector Dot:" << endl;
    c = VectorDotGPU(d_u, d_v, N);
    cout << c << endl;

    cout << endl << "MatrixVector Mult:" << endl;
    MatrixVectorMultGPU(d_A, M, N, d_x, N, d_y);
    hipMemcpy(y, d_y, N * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        cout << y[i] << endl;
    }

    cout << endl << "MatrixMatrix Mult:" << endl;
    MatrixMatrixMultGPU(d_A, M, N, d_A, M, N, d_C);
    hipMemcpy(C, d_C, N * M * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cout << C[(i * N) + j] << " ";
        }
        cout << endl;
    }

    hipFree(d_A);
    hipFree(d_C);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_x);
    hipFree(d_y);

    cout << endl << "ConjGrad:" << endl;   
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            if (i == j) { 
                A[(i*N) + j] = 5; 
            } else { 
                A[(i*N) + j] = 0; 
            } 
        }
    }

    for (int i = 0; i < M; i++) {
        x[i] = 0;
        b[i] = i * i;
    } 

    ConjugateGradient(A, M, N, b, x, 5, 0.01);
    for (int i = 0; i < M; i++) {
        cout << x[i] << endl;
    }


}
